#include "hip/hip_runtime.h"
   #include <stdio.h>
 2
 3 #define N 1000
 4
 5 //
 6 // A function marked __global__
 7 // runs on the GPU but can be called from
 8 // the CPU.
 9 //
10 // This function multiplies the elements of an array
11 // of ints by 2.
12 //
13 // The entire computation can be thought of as running
14 // with one thread per array element with blockIdx.x
15 // identifying the thread.
16 //
17 // The comparison i<N is because often it isn't convenient
18 // to have an exact 1-1 correspondence between threads
19 // and array elements. Not strictly necessary here.
20 //
21 // Note how we're mixing GPU and CPU code in the same source
22 // file. An alternative way to use CUDA is to keep
23 // C/C++ code separate from CUDA code and dynamically
24 // compile and load the CUDA code at runtime, a little
25 // like how you compile and load OpenGL shaders from
26 // C/C++ code.
27 //
28 __global__
29 void add(int *a, int *b) {
30     int i = blockIdx.x;
31     if (i<N) {
32         b[i] = 2*a[i];
33     }
34 }
35
36 int main() {
37     //
38     // Create int arrays on the CPU.
39     // ('h' stands for "host".)
40     //
41     int ha[N], hb[N];
42
43     //
44     // Create corresponding int arrays on the GPU.
45     // ('d' stands for "device".)
46     //
47     int *da, *db;
48     hipMalloc((void **)&da, N*sizeof(int));
49     hipMalloc((void **)&db, N*sizeof(int));
50
51     //
52     // Initialise the input data on the CPU.
53     //
54     for (int i = 0; i<N; ++i) {
55         ha[i] = i;
56     }
57
58     //
59     // Copy input data to array on GPU.
60     //
61     hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
62
63     //
64     // Launch GPU code with N threads, one per
65     // array element.
66     //
67     add<<<N, 1>>>(da, db);
68
69     //
70     // Copy output array from GPU back to CPU.
71     //
72     hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
73
74     for (int i = 0; i<N; ++i) {
75         printf("%d\n", hb[i]);
76     }
77
78     //
79     // Free up the arrays on the GPU.
80     //
81     hipFree(da);
82     hipFree(db);
83
84     return 0;
85 }
