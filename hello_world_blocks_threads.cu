#include "hip/hip_runtime.h"
 1 #include <stdio.h>
 2
 3 __global__ void hello_cuda()
 4 {
 5     printf("Hello CUDA world \n");
 6 }
 7
 8 int main()
 9 {
10     int nx, ny;
11     nx = 2;
12     ny = 2;
13
14     dim3 block(2, 1);
15     dim3 grid(nx / block.x,ny / block.y);
16
17     hello_cuda << < 2,2 >> > ();
18     hipDeviceSynchronize();
19
20     hipDeviceReset();
21     return 0;
22 }
23
