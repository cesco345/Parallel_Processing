#include "hip/hip_runtime.h"
   // Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
 2 // Assigns every element in an array with its index.
 3
 4 // nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple
 5
 6 #include <stdio.h>
 7
 8 const int N = 16;
 9 const int blocksize = 16;
10
11 __global__
12 void simple(float *c)
13 {
14     c[threadIdx.x] = threadIdx.x;
15 }
16
17 int main()
18 {
19     float *c = new float[N];
20     float *cd;
21     const int size = N*sizeof(float);
22
23     hipMalloc( (void**)&cd, size );
24     dim3 dimBlock( blocksize, 1 );
25     dim3 dimGrid( 1, 1 );
26     simple<<<dimGrid, dimBlock>>>(cd);
27     hipDeviceSynchronize();
28     hipMemcpy( c, cd, size, hipMemcpyDeviceToHost );
29     hipFree( cd );
30
31     for (int i = 0; i < N; i++)
32         printf("%f ", c[i]);
33     printf("\n");
34     delete[] c;
35     printf("done\n");
36     return EXIT_SUCCESS;
37 }
