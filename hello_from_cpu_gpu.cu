#include "hip/hip_runtime.h"
   // A simple Hello World CUDA program.
 2
 3 // #include the entire body of the cuPrintf code
 4 //#include "util/cuPrintf.cu"
 5
 6 // #include <stdio.h> for host printf
 7 #include <stdio.h>
 8
 9
10 __global__ void device_greetings(void)
11 {
12   printf("Hello, world from the device!\n");
13 }
14
15
16 int main(void)
17 {
18   // greet from the host
19   printf("Hello, world from the host!\n");
20
21   // initialize cuPrintf
22   //cudaPrintfInit();
23
24   // launch a kernel with a single thread to greet from th>
25   device_greetings<<<1,1>>>();
26
27   // display the device's greeting
28   hipDeviceSynchronize();
29
30   //hipDeviceReset;
31   return 0;
32 }
